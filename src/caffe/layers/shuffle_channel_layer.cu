#include "hip/hip_runtime.h"
#include "caffe/layers/shuffle_channel_layer.hpp"

namespace caffe {
template <typename Dtype>
__global__ void kernel_shuffle_channel(const int num,
        int channels) {};

template <typename Dtype>
void ShuffleChannelLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
        const vector<Blob<Dtype>*>& top) {
    //TODO 
    this->Forward_cpu(bottom, top);
}

template <typename Dtype>
void ShuffleChannelLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, 
        const vector<bool>& propagate_down, 
        const vector<Blob<Dtype>*>& bottom) {
    NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(ShuffleChannelLayer);
}  // namespace caffe
